#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
namespace cg = cooperative_groups;

// #define THREADS_PER_BLOCK 32
#define TILE_SIZE 16
#define THREADS_PER_BLOCK_X TILE_SIZE
#define THREADS_PER_BLOCK_Y TILE_SIZE
#define THREADS_PER_BLOCK (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)

#define TOL 1e-5
#define ITER_LIMIT 1000

int nthreads, n;

__device__ float diff;

__global__ void init_kernel(float *A, hiprandState_t *states, unsigned int seed, int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= (n + 2) * (n + 2)) {
        return;
    }
    hiprand_init(seed, id, 0, &states[id]);
    A[id] = hiprand_uniform(&states[id]);
}

__global__ void solve(float *A, int n, int span) {
    int done = 0, iters = 0;
    float temp, local_diff;
    cg::grid_group grid = cg::this_grid();
    while (!done) {
        local_diff = 0.0;
        if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
            diff = 0.0;
        }

        grid.sync();
        int outerTileIdx = blockIdx.x;
        int outerTileIdy = blockIdx.y;
        // printf("i: %d , j: %d \n", outerTileIdx, outerTileIdy);
        // __shared__ float as[TILE_SIZE][TILE_SIZE];

        for (int k = 0; k < span; k++) {
            for (int l = 0; l < span; l++) {
                int tileIdx = outerTileIdx * span + k;
                int tileIdy = outerTileIdy * span + l;
                int idx = tileIdx * THREADS_PER_BLOCK_X + threadIdx.x + 1;
                int idy = tileIdy * THREADS_PER_BLOCK_Y + threadIdx.y + 1;
                // printf("i: %d , j: %d \n", tileIdx, tileIdy);

                // printf("i: %d , j: %d \n", idx, idy);
                // as[threadIdx.x][threadIdx.y] = A[idx * n + idy];
                temp = A[idx * n + idy];
                // grid.sync();
                A[idx * n + idy] = 0.2 * (A[idx * n + idy] + A[idx * n + idy - 1] + A[idx * n + idy + 1] + A[(idx + 1) * n + idy] + A[(idx - 1) * n + idy]);
                local_diff += fabs(A[idx * n + idy] - temp);
            }
        }
        // int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // int idy = blockIdx.y * blockDim.y + threadIdx.y;
        // int id = idx + idy * (gridDim.x * blockDim.x) * idy;
        // unsigned mask = 0xffffffff;
        // for (int i = warpSize / 2; i > 0; i = i / 2) {
        //     local_diff += __shfl_down_sync(mask, local_diff, i);
        // }
        // if (id % warpSize == 0) {
        //     atomicAdd(&diff, local_diff);
        // }
        atomicAdd(&diff, local_diff);

        grid.sync();
        iters++;

        if (((diff / (n * n) < TOL) || (iters == ITER_LIMIT))) {
            done = 1;
        }

        if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
            printf("[%d] diff = %.10f\n", iters, diff / (n * n));
        }
        grid.sync();
    }
}

int main(int argc, char **argv) {
    struct timeval tv0, tv1;
    struct timezone tz0, tz1;
    float *A;

    // if (argc != 3) {
    //     printf("Need grid size (n) and number of threads (nthreads).\nAborting...\n");
    //     exit(1);
    // }

    n = atoi(argv[1]);
    nthreads = atoi(argv[2]);
    // n = 1024;
    // nthreads = 1024;

    hipMallocManaged((void **)&A, sizeof(float) * (n + 2) * (n + 2));

    int numBlocksPerSm = 0, numBlocks;
    hipDeviceProp_t deviceProp;
    int device = -1;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, solve, THREADS_PER_BLOCK, 0);
    numBlocks = deviceProp.multiProcessorCount * numBlocksPerSm;
    printf("Max number of blocks per SM: %d, number of SMs: %d, number of blocks: %d\n", numBlocksPerSm, deviceProp.multiProcessorCount, numBlocks);
    while ((numBlocks & (numBlocks - 1)) != 0) numBlocks--;
    if (nthreads > (THREADS_PER_BLOCK * numBlocks)) nthreads = THREADS_PER_BLOCK * numBlocks;
    printf("Number of blocks: %d, Threads per block: %d, Total number of threads: %d\n", nthreads / THREADS_PER_BLOCK, THREADS_PER_BLOCK, nthreads);
    // return 0;
    nthreads = round(sqrt(nthreads));
    printf("nthreads: %d\n", nthreads);

    hiprandState_t *states;
    hipMalloc((void **)&states, (n + 2) * (n + 2) * sizeof(hiprandState_t));

    init_kernel<<<(((n + 2) * (n + 2)) / 1024) + ((n + 2) * (n + 2)) % 1024, 1024>>>(A, states, time(0), n);
    hipDeviceSynchronize();

    int num_elements_per_thread = n / nthreads;
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, device);
    void *kernelArgs[] = {(void *)&A, (void *)&n, (void *)&num_elements_per_thread};
    // dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
    // dim3 dimGrid(nthreads / THREADS_PER_BLOCK, 1, 1);
    dim3 dimBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 dimGrid(nthreads / THREADS_PER_BLOCK_X, nthreads / THREADS_PER_BLOCK_Y);

    gettimeofday(&tv0, &tz0);
    hipLaunchCooperativeKernel((void *)solve, dimGrid, dimBlock, kernelArgs);
    hipDeviceSynchronize();

    gettimeofday(&tv1, &tz1);
    hipError_t err = hipGetLastError();  // Get error code

    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    printf("[Main] Done!\n");

    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%f ", A[i * n + j]);
    //     }
    //     printf("\n");
    //     // break;
    // }

    printf("Time: %lf seconds\n", (double)((tv1.tv_sec - tv0.tv_sec) * 1000000 + (tv1.tv_usec - tv0.tv_usec)) / 1000000);

    return 0;
}
